#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "schur_opt_gpu.h"

using namespace std;

void SchurOpt::from_g2o(/* parameters */) {

}

void SchurOpt::to_g2o(/* parameters */) {

}

inline int pair_to_idx(const int r, const int c, const int max_r, const int max_c) {
    return r * max_c + c;
}


void SchurOpt::read_sparse(const string& fname, SchurOpt& schur_opt, WhichBlock which_block) {
    ifstream fin(fname);

    if(!fin.is_open()) {
        cerr << "Error opening: " << fname << endl;
    }

    string line;
    getline(fin, line); // first 3 are unimportant
    getline(fin, line);
    getline(fin, line);

    int num_rows, num_cols;
    string garbage;

    fin >> garbage >> garbage >> num_rows >> garbage >> garbage >> num_cols;

    // cout << "num_rows = " << num_rows << " num_cols = " << num_cols << endl;

    assert(num_rows % 3 == 0);
    assert(num_cols % 3 == 0);

    int row, col;
    double val;

    if(which_block == WhichBlock::isA) {
        int num_blocks = num_rows / block_size;

        A_sparse = vector<vector<double>>(num_blocks, vector<double>(block_squared, 0));
        L = num_blocks * block_size;

        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(abs(row - col) < 3);

            int block_id = row / 3;
            assert(block_id < num_blocks);

            int i_offset = row % 3, j_offset = col % 3;
            A_sparse[block_id][pair_to_idx(i_offset, j_offset, block_size, block_size)] = val;
        }
    } else if (which_block == WhichBlock::isB || which_block == WhichBlock::isC){
        if(which_block == WhichBlock::isC) {
            // swap max rows and cols because we're reading the transpose
            int temp = num_rows;
            num_rows = num_cols;
            num_cols = temp;
        }
        P = num_cols;
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        B = vector<vector<double>>(num_row_blocks * num_col_blocks, vector<double>(block_squared, 0));
        B_used = vector<bool>(num_row_blocks * num_col_blocks, false);
        
        while(fin >> row >> col >> val) {
            if(which_block == WhichBlock::isC) {
                // Swap col and row because we're reading the transpose
                int temp = row;
                row = col;
                col = temp;
            }

            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int row_block = row / block_size;
            int col_block = col / block_size;

            int block_idx = pair_to_idx(row_block, col_block, num_row_blocks, num_col_blocks);
            int i_offset = row % block_size;
            int j_offset = col % block_size;
            int idx = pair_to_idx(i_offset, j_offset, block_size, block_size);
            B[block_idx][idx] = val;
            B_used[block_idx] = true;

            // cout << row_block << " " << col_block << " " << block_idx << " " << i_offset << " " << j_offset << " " << val << endl;
        }
    } else if (which_block == WhichBlock::isD){
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        cout << "read D" << num_row_blocks << " " << num_col_blocks << endl;

        D = vector<vector<double>>(num_row_blocks * num_col_blocks, vector<double>(block_squared, 0));
        D_used = vector<bool>(num_row_blocks * num_col_blocks, false);
        
        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int row_block = row / block_size;
            int col_block = col / block_size;

            int block_idx = pair_to_idx(row_block, col_block, num_row_blocks, num_col_blocks);
            int i_offset = row % block_size;
            int j_offset = col % block_size;
            int idx = pair_to_idx(i_offset, j_offset, block_size, block_size);

            D[block_idx][idx] = val;
            D_used[block_idx] = true;
        }
    }
}

void SchurOpt::compute_schur(/* parameters */) {
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    double* A_gpu;
    cudaStat = hipMalloc((void**) &A_gpu, block_squared * A_sparse.size());
    cout << cudaStat << endl;
}
