#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "schur_opt_gpu.h"
#include <chrono>

using namespace std;

void SchurOpt::from_g2o(/* parameters */) {

}

void SchurOpt::to_g2o(/* parameters */) {

}

SchurOpt::~SchurOpt() {
}

// For CUBLAS, this needs to be column major
inline int pair_to_idx_col(const int r, const int c, const int max_r, const int max_c) {
    return r + c * max_r;
}

inline int pair_to_idx_row(const int r, const int c, const int max_r, const int max_c) {
    return c + r * max_c;
}

void print_error_and_exit(const string& msg) {
    cout << msg << endl;
    exit(1);
}


/*void SchurOpt::read_sparse(const string& fname, WhichBlock which_block) {
    ifstream fin(fname);

    if(!fin.is_open()) {
        cerr << "Error opening: " << fname << endl;
        exit(1);
    }

    string line;
    getline(fin, line); // first 3 are unimportant
    getline(fin, line);
    getline(fin, line);

    int num_rows, num_cols;
    string garbage;

    fin >> garbage >> garbage >> num_rows >> garbage >> garbage >> num_cols;

    // cout << "num_rows = " << num_rows << " num_cols = " << num_cols << endl;

    assert(num_rows % 3 == 0);
    assert(num_cols % 3 == 0);

    int row, col;
    double val;

    if(which_block == WhichBlock::isA) {
        int num_blocks = num_rows / block_size;

        A_sparse = vector<double>(num_blocks * block_squared, 0);
        L = num_blocks * block_size;

        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(abs(row - col) < 3);

            int block_id = row / 3;
            assert(block_id < num_blocks);

            int i_offset = row % 3, j_offset = col % 3;
            A_sparse[block_id * block_squared + pair_to_idx(i_offset, j_offset, block_size, block_size)] = val;
        }
    } else if (which_block == WhichBlock::isB || which_block == WhichBlock::isC){
        if(which_block == WhichBlock::isC) {
            // swap max rows and cols because we're reading the transpose
            int temp = num_rows;
            num_rows = num_cols;
            num_cols = temp;
        }
        P = num_cols;
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        B = vector<double>(num_row_blocks * num_col_blocks * block_squared, 0);
        B_used = vector<bool>(num_row_blocks * num_col_blocks, false);
        
        while(fin >> row >> col >> val) {
            if(which_block == WhichBlock::isC) {
                // Swap col and row because we're reading the transpose
                int temp = row;
                row = col;
                col = temp;
            }

            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int row_block = row / block_size;
            int col_block = col / block_size;

            int block_idx = pair_to_idx(row_block, col_block, num_row_blocks, num_col_blocks);
            int i_offset = row % block_size;
            int j_offset = col % block_size;
            int idx = pair_to_idx(i_offset, j_offset, block_size, block_size);
            B[block_idx * block_squared + idx] = val;
            B_used[block_idx] = true;

            // cout << row_block << " " << col_block << " " << block_idx << " " << i_offset << " " << j_offset << " " << val << endl;
        }
    } else if (which_block == WhichBlock::isD){
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        cout << "read D" << num_row_blocks << " " << num_col_blocks << endl;

        D = vector<vector<double>>(num_row_blocks * num_col_blocks, vector<double>(block_squared, 0));
        D_used = vector<bool>(num_row_blocks * num_col_blocks, false);
        
        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int row_block = row / block_size;
            int col_block = col / block_size;

            int block_idx = pair_to_idx(row_block, col_block, num_row_blocks, num_col_blocks);
            int i_offset = row % block_size;
            int j_offset = col % block_size;
            int idx = pair_to_idx(i_offset, j_offset, block_size, block_size);

            D[block_idx][idx] = val;
            D_used[block_idx] = true;
        }
    }
}*/

void SchurOpt::read_sparse(const string& fname, WhichBlock which_block) {
    ifstream fin(fname);

    if(!fin.is_open()) {
        cerr << "Error opening: " << fname << endl;
        exit(1);
    }

    string line;
    getline(fin, line); // first 3 are unimportant
    getline(fin, line);
    getline(fin, line);

    int num_rows, num_cols;
    string garbage;

    fin >> garbage >> garbage >> num_rows >> garbage >> garbage >> num_cols;

    // cout << "num_rows = " << num_rows << " num_cols = " << num_cols << endl;

    assert(num_rows % 3 == 0);
    assert(num_cols % 3 == 0);

    int row, col;
    double val;

    if(which_block == WhichBlock::isA) {
        int num_blocks = num_rows / block_size;

        A_sparse = vector<double>(num_blocks * block_squared, 0);
        L = num_blocks * block_size;

        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(abs(row - col) < 3);

            int block_id = row / 3;
            assert(block_id < num_blocks);

            int i_offset = row % 3, j_offset = col % 3;
            A_sparse[block_id * block_squared + pair_to_idx_col(i_offset, j_offset, block_size, block_size)] = val;
        }
    } else if (which_block == WhichBlock::isB || which_block == WhichBlock::isC){
        if(which_block == WhichBlock::isB) {
            // swap max rows and cols because we're reading the transpose
            int temp = num_rows;
            num_rows = num_cols;
            num_cols = temp;
        }
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        C = vector<double>(num_row_blocks * num_col_blocks * block_squared, 0);
        
        while(fin >> row >> col >> val) {
            if(which_block == WhichBlock::isB) {
                // Swap col and row because we're reading the transpose
                int temp = row;
                row = col;
                col = temp;
            }

            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int idx = pair_to_idx_col(row, col, num_rows, num_cols);
            C[idx] = val;

            // cout << num_rows << " " << num_cols << " " << row << " " << col << " " << idx << " " << val << endl;
        }
    } else if (which_block == WhichBlock::isD){
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        P = num_rows;
        // cout << "read D" << num_row_blocks << " " << num_col_blocks << endl;

        D = vector<double>(num_row_blocks * num_col_blocks * block_squared, 0);
        
        while(fin >> row >> col >> val) {
            row--;  // index by 1
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int idx = pair_to_idx_col(row, col, num_rows, num_cols);
            D[idx] = val;
        }
    } else if (which_block == WhichBlock::isDschur_ref){
        int num_row_blocks = num_rows / block_size;
        int num_col_blocks = num_cols / block_size;

        Dschur_ref = vector<double>(num_row_blocks * num_col_blocks * block_squared, 0);
        
        while(fin >> row >> col >> val) {
            row--; // decrease index by 1 as we are zero indexed
            col--;

            assert(row < num_rows);
            assert(col < num_cols);

            int idx = pair_to_idx_col(row, col, num_rows, num_cols);

            Dschur_ref[idx] = val;
        }
    }
}

/**
 * Dschur - Schur matrix the solver calculated
 * Dschur_ref - Schur matrix the G2O block solver outputted
 * Compare dimension, sparisty, and calculate the MSE. 
 */
void SchurOpt::verify_correctness(/* parameters */) {

    // If you set this, MSE will be 0
    // Dschur_ref = Dschur;
    // Dschur_ref_used = Dschur_used;

    // verify Dschur has the right size
    assert(Dschur.size() == P * P);
    
    // actual code
    assert(Dschur.size() == Dschur_ref.size()); // comparing number of 3x3 blocks

    // cout << "Dschur size=" << Dschur.size() << endl;
    double se = 0.0;  // squared error
    for(int i = 0; i < Dschur.size(); i++) {
        double diff = Dschur[i] - Dschur_ref[i];
        se += diff * diff;
    }
    double mse = se / (double) (P*P);
    cout << "MSE: " << mse << endl;
}

void SchurOpt::mem_alloc() {
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS initialization failed");
    }

    cudaStat = hipMalloc((void**) &A_gpu, A_sparse.size() * sizeof(double));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure");
    }
    stat = hipblasSetVector(A_sparse.size(), sizeof(double), A_sparse.data(), 1, A_gpu, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed");
    }

    cudaStat = hipMalloc((void**) &Ainv_gpu, A_sparse.size() * sizeof(double));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure: Aing_gpu");
    }

    assert(A_sparse.size() % block_squared == 0);
    batch_size = A_sparse.size() / block_squared;

    cudaStat = hipMalloc((void**) &A_gpu_batch, batch_size * sizeof(double*));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure for A_gpu_batch");
    }

    cudaStat = hipMalloc((void**) &Ainv_gpu_batch, batch_size * sizeof(double*));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure for Ainv_gpu_batch");
    }
    
    vector<double*> A_batch(batch_size, nullptr);
    vector<double*> Ainv_batch(batch_size, nullptr);
    for(int i = 0; i < batch_size; i++) {
        A_batch[i] = A_gpu + i * block_squared;
        Ainv_batch[i] = Ainv_gpu + i * block_squared;
    }

    stat = hipblasSetVector(batch_size, sizeof(double*), A_batch.data(), 1, A_gpu_batch, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: A_gpu_batch");
    }

    stat = hipblasSetVector(batch_size, sizeof(double*), Ainv_batch.data(), 1, Ainv_gpu_batch, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: Ainv_gpu_batch");
    }

    cudaStat = hipMalloc((void**) &info_gpu, batch_size * sizeof(int));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure: info_gpu");
    }

    cudaStat = hipMalloc((void**) &C_gpu, C.size() * sizeof(double));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure: C_gpu");
    }
    stat = hipblasSetVector(C.size(), sizeof(double), C.data(), 1, C_gpu, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: C_gpu");
    }

    cudaStat = hipMalloc((void**) &CAinv_gpu, C.size() * sizeof(double));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure: CAinv_gpu");
    }

    cudaStat = hipMalloc((void**) &C_gpu_batch, batch_size * sizeof(double*));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure for C_gpu_batch");
    }

    cudaStat = hipMalloc((void**) &CAinv_gpu_batch, batch_size * sizeof(double*));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure for CAinv_gpu_batch");
    }

    int panel_size = block_size * P;

    vector<double*> C_batch(batch_size, nullptr);
    vector<double*> CAinv_batch(batch_size, nullptr);
    for(int i = 0; i < batch_size; i++) {
        C_batch[i] = C_gpu + i * panel_size;
        CAinv_batch[i] = CAinv_gpu + i * panel_size;
    }

    stat = hipblasSetVector(batch_size, sizeof(double*), C_batch.data(), 1, C_gpu_batch, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: C_gpu_batch");
    }

    stat = hipblasSetVector(batch_size, sizeof(double*), CAinv_batch.data(), 1, CAinv_gpu_batch, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: Ainv_gpu_batch");
    }

    cudaStat = hipMalloc((void**) &D_gpu, D.size() * sizeof(double*));
    if(cudaStat != hipSuccess) {
        print_error_and_exit("Error: Device memory allocation failure for D_gpu");
    }

    stat = hipblasSetVector(D.size(), sizeof(double), D.data(), 1, D_gpu, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS vector copy failed: D_gpu");
    }

}

void SchurOpt::mem_dealloc() {
    if(A_gpu) {
        hipFree(A_gpu);
        A_gpu = nullptr;
    }
    if(Ainv_gpu) {
        hipFree(Ainv_gpu);
        Ainv_gpu = nullptr;
    }
    if(A_gpu_batch) {
        hipFree(A_gpu_batch);
        A_gpu_batch = nullptr;
    }
    if(Ainv_gpu_batch) {
        hipFree(Ainv_gpu_batch);
        Ainv_gpu_batch = nullptr;
    }
    if(B_gpu) {
        hipFree(B_gpu);
        B_gpu = nullptr;
    }
    if(B_gpu_batch) {
        hipFree(B_gpu_batch);
        B_gpu_batch = nullptr;
    }
    if(C_gpu) {
        hipFree(C_gpu);
        C_gpu = nullptr;
    }
    if(C_gpu_batch) {
        hipFree(C_gpu_batch);
        C_gpu_batch = nullptr;
    }
    if(CAinv_gpu) {
        hipFree(CAinv_gpu);
        CAinv_gpu = nullptr;
    }
    if(CAinv_gpu_batch) {
        hipFree(CAinv_gpu_batch);
        CAinv_gpu_batch = nullptr;
    }
    if(D_gpu) {
        hipFree(D_gpu);
        D_gpu = nullptr;
    }
    if(info_gpu) {
        hipFree(info_gpu);
        info_gpu = nullptr;
    }
    hipblasDestroy(handle);
}

void SchurOpt::compute_Ainv() {


    stat = cublasDmatinvBatched(handle, 3, A_gpu_batch, 3, Ainv_gpu_batch, 3, info_gpu, batch_size);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("Error: Inverting A");
    }

    Ainv = vector<double>(A_sparse.size(), 0);
    stat = hipblasGetVector(A_sparse.size(), sizeof(double), Ainv_gpu, 1, Ainv.data(), 1);   
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("Error: Getting vector");
    }

}

void SchurOpt::compute_schur(/* parameters */) {
    mem_alloc();

    std::chrono::steady_clock::time_point t_schur_start = std::chrono::steady_clock::now();

    compute_Ainv();

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasOperation_t transt = HIPBLAS_OP_T;


    double alpha = 1, beta = 0;

    stat = hipblasDgemmBatched(handle, 
                              transa, 
                              transb, 
                              P, block_size, block_size, 
                              &alpha, 
                              C_gpu_batch, P, 
                              Ainv_gpu_batch, block_size, 
                              &beta, 
                              CAinv_gpu_batch, P, 
                              batch_size);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("CUBLAS batched dgemm failed");
    }

    // vector<double> CAinv(C.size(), 0);
    // stat = hipblasGetVector(C.size(), sizeof(double), CAinv_gpu, 1, CAinv.data(), 1);   
    // if(stat != HIPBLAS_STATUS_SUCCESS) {
    //     print_error_and_exit("Error getting data: CAinv");
    // }

    // cout << "CAinv: " << endl;
    // for(int i = 0; i < 100; i++) {
    //     cout << CAinv[i] << endl;
    // }

    alpha = -1; // D - CAinvB
    beta = 1;

    stat = hipblasDgemm(handle,
                       transa,
                       transt,
                       P, P, L,
                       &alpha,
                       CAinv_gpu, P,
                       C_gpu, P,
                       &beta,
                       D_gpu, P);

    Dschur = vector<double>(D.size(), 0);
    stat = hipblasGetVector(Dschur.size(), sizeof(double), D_gpu, 1, Dschur.data(), 1);   
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        print_error_and_exit("Error getting data: Dschur");
    }

    chrono::steady_clock::time_point t_schur_end = chrono::steady_clock::now();
    double t_schur = chrono::duration_cast<chrono::duration<double, milli>>(t_schur_end - t_schur_start).count();

    cout << "[STATS] "  << "t_schur= " << t_schur << " ms" << endl;


    // cout << "Dschur: " << endl;
    // for(int i = 0; i < 200; i++) {
    //     cout << Dschur[i] << endl;
    // }

    mem_dealloc();
}
